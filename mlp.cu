#include "hip/hip_runtime.h"
#include "mlp.h"
#include "linalg.h"
#include "layer.h"
#include "common.h"
#include <stdio.h>


__device__ void apply_sigmoid(vector_t vector) {
    f32 *val = &vector.vals[threadIdx.x];
    *val = 1 / (1 + expf(-*val));
}

__device__ void apply_softmax(vector_t vector) {
    // TODO: Optimize
    f32 *val = &vector.vals[threadIdx.x];
    *val = expf(*val);
    __syncthreads();
    f32 sum = 0;
    for (u32 i = 0; i != vector.size; i++) {
        sum += vector.vals[i];
    }
    *val /= sum;
}


__global__ void layer_compute(layer_t layer, vector_t in_vector, vector_t out_vector) {
    ASSERT_EQ_INT(layer.in_dimension, in_vector.size);
    ASSERT_EQ_INT(layer.out_dimension, out_vector.size);
    matrix_t weights = layer.weights;
    vector_t bias = layer.bias;

    // out_vector = weights * in_vector + bias
    matrix_vector_multiply(weights, in_vector, out_vector);
    out_vector.vals[threadIdx.x] += bias.vals[threadIdx.x];

    switch (layer.activation_func) {
        case SOFTMAX:
            apply_softmax(out_vector);
            break;
        case SIGMOID:
            apply_sigmoid(out_vector);
            break;
    }
}


void network_compute(mlp_t *mlp, vector_t *layer_outputs, vector_t *in_vector) {
    // Compute output of each layer
    for (int i = 0; i != mlp->num_layers; i++) {
        vector_t *layer_in_vec = (i == 0) ? in_vector : &layer_outputs[i-1];
        vector_t *layer_out_vec = &layer_outputs[i];
        layer_t *layer = &mlp->layers[i];
        layer_compute<<<1, layer->out_dimension>>>(*layer, *layer_in_vec, *layer_out_vec);
        hipDeviceSynchronize();
    }
}


// mlp: host-located. mlp.layers is also host-located data
// gradient: host-located
// layer_outputs: host-located array of vectors, where vector-data is on device
// in_vector: host-located vector where data is on device
// expected_out_vector: host-located vector where data is on device
void compute_gradient(mlp_t *mlp, mlp_t *gradient, vector_t *layer_outputs, vector_t *in_vector, vector_t *expected_out_vector) {
    network_compute(mlp, layer_outputs, in_vector);
    // Compute derivative with respect to output layer dJ/dy(L)
    // vector_t *mlp_output = layer_outputs[mlp->num_layers];
    // cost_function_derivative(mlp->cost_func, mlp_output, expected_out_vector, )
}


vector_t *alloc_layer_outputs(mlp_t *mlp) {
    vector_t *vectors = (vector_t *) malloc(sizeof(vector_t) * mlp->num_layers);
    ASSERT_NOT_NULL(vectors);
    for (u32 i = 0; i != mlp->num_layers; i++) {
        vector_init(&vectors[i], mlp->layers[i].out_dimension);
    }

    return vectors;
}


int main() {

    // Values in layers
    f32 layer_0_bias_raw[5] = {
        -2, -1, 0, 1, 2
    };
    f32 layer_1_bias_raw[3] = {
        2, 2, 2
    };
    vector_t layer_0_bias = {
        .size = 5,
        .vals = layer_0_bias_raw
    };
    vector_t layer_1_bias = {
        .size = 3,
        .vals = layer_1_bias_raw
    };

    mlp_t mlp = {
        .num_layers = 2,
        .layers = (layer_t *) malloc(sizeof(layer_t) * 2),
        .cost_func = CROSS_ENTROPY
    };

    // Create input vector and mlp
    vector_t input;
    vector_init(&input, 2);
    layer_init(&mlp.layers[0], 2, 5, SIGMOID);
    layer_init(&mlp.layers[1], 5, 3, SOFTMAX);
    vector_t *layer_outputs = alloc_layer_outputs(&mlp);

    // Move bias vectors into MLP layers
    vector_host_to_device(&mlp.layers[0].bias, &layer_0_bias);
    vector_host_to_device(&mlp.layers[1].bias, &layer_1_bias);

    // Compute
    network_compute(&mlp, layer_outputs, &input);

    // Alloc memory to get data from device
    vector_t host_layer_outputs[2] = {
        {
            .size = 5,
            .vals = (f32 *) malloc(sizeof(f32) * 10)
        },
        {
            .size = 3,
            .vals = (f32 *) malloc(sizeof(f32) * 10)
        }
    };

    for (i32 i = 0; i != 10; i++) {
        host_layer_outputs[0].vals[i] = -2*i;
        host_layer_outputs[1].vals[i] = -2*i;
    }

    vector_device_to_host(&host_layer_outputs[0], &layer_outputs[0]);
    for (i32 i = 0; i != 10; i++)
        printf("layers 0 [%d] = %f\n", i, host_layer_outputs[0].vals[i]);

    vector_device_to_host(&host_layer_outputs[1], &layer_outputs[1]);
    for (i32 i = 0; i != 10; i++)
        printf("layers 1 [%d] = %f\n", i, host_layer_outputs[1].vals[i]);


}


// __global__ void cross_entropy_derivative(vector_t *estimate, vector_t *truth, vector_t *derivative_out) {

// }




// __global__ void layer_


// __global__ void layer_back_propagate_step(layer_t *layer, vector_t *y_self, vector_t *y_minus, )