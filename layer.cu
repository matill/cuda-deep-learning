#include "hip/hip_runtime.h"
#include "layer.h"
#include "common.h"
#include "linalg.h"


void layer_init(layer_t *layer, u32 in_dimension, u32 out_dimension, activation_func_t activation_func) {
    layer->in_dimension = in_dimension;
    layer->out_dimension = out_dimension;
    layer->activation_func = activation_func;
    matrix_init(&layer->weights, out_dimension, in_dimension);
    vector_init(&layer->bias, out_dimension);
}


__device__ void apply_sigmoid(vector_t vector) {
    f32 *val = &vector.vals[threadIdx.x];
    *val = 1 / (1 + expf(-*val));
}


__device__ void apply_softmax(vector_t vector) {
    // TODO: Optimize
    f32 *val = &vector.vals[threadIdx.x];
    *val = expf(*val);
    __syncthreads();
    f32 sum = 0;
    for (u32 i = 0; i != vector.size; i++) {
        sum += vector.vals[i];
    }
    *val /= sum;
}


__global__ void layer_compute(layer_t layer, vector_t in_vector, vector_t out_vector) {
    ASSERT_EQ_INT(layer.in_dimension, in_vector.size);
    ASSERT_EQ_INT(layer.out_dimension, out_vector.size);
    matrix_t weights = layer.weights;
    vector_t bias = layer.bias;

    // out_vector = weights * in_vector + bias
    matrix_vector_multiply(weights, in_vector, out_vector);
    out_vector.vals[threadIdx.x] += bias.vals[threadIdx.x];

    switch (layer.activation_func) {
        case SOFTMAX:
            apply_softmax(out_vector);
            break;
        case SIGMOID:
            apply_sigmoid(out_vector);
            break;
    }
}


__global__ void compute_gradient_v_plus_to_v(vector_t v_gradient, vector_t v_plus_gradient, matrix_t w_plus, vector_t y, activation_func_t activation_func) {
    // Compute dJ / dy(r)
    u32 k = threadIdx.x;
    f32 y_k_derivarive = 0;
    u32 num_iters = w_plus.height;
    for (u32 i = 0; i != num_iters; i++) {
        y_k_derivarive += v_plus_gradient.vals[i] * matrix_index(w_plus, i, k);
    }

    // Compute dJ / dv(r) according to activation function
    f32 v_k_derivative;
    f32 y_k = y.vals[k];
    switch (activation_func) {
        case SIGMOID:
            v_k_derivative = y_k * (y_k - 1) * y_k_derivarive;
            break;

        case SOFTMAX:
            // Temporarily store all y_k_derivative in shared memory to compute sigma_c
            // TODO: More efficient (parallel and shared) computation of sigma_c
            v_gradient.vals[k] = y_k_derivarive;
            __syncthreads();
            f32 sigma_c = vector_dot(v_gradient, y);

            // Compute v derivatives according to formulas. Sync to avoid modifying v_gradient buffer before all threads are done using it
            __syncthreads();
            v_k_derivative = y_k * (y_k_derivative  - sigma_c);
            break;
    }

    // Store derivatives in output vector
    v_gradient.vals[k] = v_k_derivative;
}


// Call with
// gridDim.x = layer.out_dim
// blockDim.x = layer.in_dim
__global__ void compute_weight_gradient(matrix_t w_derivative_out, vector_t v_gradient, vector_t y_minus) {
    u32 i = blockIdx.x;
    u32 j = gridDim.x;
    f32 v_gradient_i = v_gradient.vals[i];
    f32 y_minus_j = y_minus.vals[j];
    *matrix_index(w_derivative_out, i, j) = v_gradient_i * y_minus_j;
}
